#include "hip/hip_runtime.h"
struct instance_params;
struct h_instance_params;

#include <pthread.h>

#include <hip/hip_runtime.h>

typedef pthread_t CUTThread;
typedef void *(*CUT_THREADROUTINE)(void *);

#define CUT_THREADPROC void
#define  CUT_THREADEND
//Create thread
CUTThread start_thread(CUT_THREADROUTINE func, void * data){
    pthread_t thread;
    pthread_create(&thread, NULL, func, data);
    return thread;
}

//Wait for thread to finish
void end_thread(CUTThread thread){
    pthread_join(thread, NULL);
}

//Destroy thread
void destroy_thread( CUTThread thread ){
    pthread_cancel(thread);
}

//Wait for multiple threads
void wait_for_threads(const CUTThread * threads, int num){
    for(int i = 0; i < num; i++)
        end_thread( threads[i] );
}



#include <stdio.h>
#include <math.h>
#include <string.h>

#include "../depends/libstl-cuda/memory.cuh"
#include "../depends/libstl-cuda/vector.cuh"
#include "../depends/libstl-cuda/utility.cuh"

#include "../depends/libff-cuda/fields/bigint_host.cuh"
#include "../depends/libff-cuda/fields/fp_host.cuh"
#include "../depends/libff-cuda/fields/fp2_host.cuh"
#include "../depends/libff-cuda/fields/fp6_3over2_host.cuh"
#include "../depends/libff-cuda/fields/fp12_2over3over2_host.cuh"
#include "../depends/libff-cuda/curves/bls12_381/bls12_381_init_host.cuh"
#include "../depends/libff-cuda/curves/bls12_381/bls12_381_g1_host.cuh"
#include "../depends/libff-cuda/curves/bls12_381/bls12_381_g2_host.cuh"
#include "../depends/libff-cuda/curves/bls12_381/bls12_381_pp_host.cuh"
#include "../depends/libmatrix-cuda/transpose/transpose_ell2csr.cuh"
#include "../depends/libmatrix-cuda/spmv/csr-balanced.cuh"
#include "../depends/libff-cuda/scalar_multiplication/multiexp.cuh"


#include "../depends/libff-cuda/curves/bls12_381/bls12_381_init.cuh"
#include "../depends/libff-cuda/curves/bls12_381/bls12_381_pp.cuh"

#include <time.h>

using namespace libff;

struct instance_params
{
    bls12_381_Fr instance;
    bls12_381_G1 g1_instance;
    bls12_381_G2 g2_instance;
    bls12_381_GT gt_instance;
};

struct h_instance_params
{
    bls12_381_Fr_host h_instance;
    bls12_381_G1_host h_g1_instance;
    bls12_381_G2_host h_g2_instance;
    bls12_381_GT_host h_gt_instance;
};


template<typename ppT>
struct MSM_params
{
    libstl::vector<libff::Fr<ppT>> vf;
    libstl::vector<libff::G1<ppT>> vg;
};


__global__ void init_params()
{
    gmp_init_allocator_();
    bls12_381_pp::init_public_params();
}

__global__ void instance_init(instance_params* ip)
{
    ip->instance = bls12_381_Fr(&bls12_381_fp_params_r);
    ip->g1_instance = bls12_381_G1(&g1_params);
    ip->g2_instance = bls12_381_G2(&g2_params);
    ip->gt_instance = bls12_381_GT(&bls12_381_fp12_params_q);
}

void instance_init_host(h_instance_params* ip)
{
    ip->h_instance = bls12_381_Fr_host(&bls12_381_fp_params_r_host);
    ip->h_g1_instance = bls12_381_G1_host(&g1_params_host);
    ip->h_g2_instance = bls12_381_G2_host(&g2_params_host);
    ip->h_gt_instance = bls12_381_GT_host(&bls12_381_fp12_params_q_host);
}
// Print memory usage statistics
void gpu_mem_status()
{
    size_t free_byte;
    size_t total_byte;

    hipError_t cuda_status = hipMemGetInfo(&free_byte, &total_byte);

    if (hipSuccess != cuda_status) {
        std::cerr << "Error: hipMemGetInfo fails, " << hipGetErrorString(cuda_status) << std::endl;
        return 1;
    }

    double free_db = (double)free_byte;
    double total_db = (double)total_byte;
    double used_db = total_db - free_db;

    std::cout << "GPU memory usage: used = " << used_db / 1024.0 / 1024.0 << " MB, free = " << free_db / 1024.0 / 1024.0 << " MB, total = " << total_db / 1024.0 / 1024.0 << " MB" << std::endl;

}

template<typename ppT>
__global__ void generate_MP(MSM_params<ppT>* mp, instance_params* ip, size_t size)
{
    new ((void*)mp) MSM_params<ppT>();
    mp->vf.presize(size, 512, 32);
    mp->vg.presize(size, 512, 32);

    libstl::launch<<<512, 32>>>
    (
        [=]
        __device__ ()
        {
            size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
            size_t tnum = blockDim.x * gridDim.x;
            libff::Fr<ppT> f = ip->instance.random_element();
            libff::G1<ppT> g = ip->g1_instance.random_element();
            f ^= idx;
            g = g * idx;
            while(idx < size)
            {
                mp->vf[idx] = f;
                mp->vg[idx] = g;
                f = f + f;
                g = g + g;
                idx += tnum;
            }
        }
    );
    hipDeviceSynchronize();

    ip->g1_instance.p_batch_to_special(mp->vg, 160, 32);
}

struct Mem
{
    size_t device_id;
    void* mem;
};

void* multi_init_params(void* params)
{
    Mem* device_mem = (Mem*) params;
    hipSetDevice(device_mem->device_id);
    gpu_mem_status();
    printf("LOG: Starting to allocating memory for Parameters ....\n");
    size_t init_size = 1024 * 1024 * 1024;
    init_size *= 12;
    if( hipMalloc( (void**)&device_mem->mem, init_size ) != hipSuccess) printf("device malloc error!\n");
    printf("LOG: Done to allocating memory for Parameters ....\n");
    libstl::initAllocator(device_mem->mem, init_size);
    init_params<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}

struct Instance
{
    size_t device_id;
    instance_params** ip;
};

void* multi_instance_init(void* instance)
{
    Instance* it = (Instance*)instance;
    hipSetDevice(it->device_id);
    if( hipMalloc( (void**)it->ip, sizeof(instance_params)) != hipSuccess) printf("ip malloc error!\n");
    instance_init<<<1, 1>>>(*it->ip);
    hipDeviceSynchronize();
    return 0;
}

template<typename ppT>
struct MSM
{
    size_t device_id;
    MSM_params<ppT>* mp;
    instance_params* ip;
    libff::G1<ppT>* res;
};

template<typename ppT>
void* multi_MSM(void* msm)
{
    MSM<ppT>* it = (MSM<ppT>*)msm;
    hipSetDevice(it->device_id);

    size_t lockMem;
    libstl::lock_host(lockMem);
    libff::p_multi_exp_faster_multi_GPU_host<libff::G1<ppT>, libff::Fr<ppT>, libff::multi_exp_method_naive_plain>(it->mp->vg, it->mp->vf, it->ip->instance, it->ip->g1_instance, 512, 32);
    hipDeviceSynchronize();
    libff::p_multi_exp_faster_multi_GPU_host<libff::G1<ppT>, libff::Fr<ppT>, libff::multi_exp_method_naive_plain>(it->mp->vg, it->mp->vf, it->ip->instance, it->ip->g1_instance, 512, 32);
    hipDeviceSynchronize();
    libff::p_multi_exp_faster_multi_GPU_host<libff::G1<ppT>, libff::Fr<ppT>, libff::multi_exp_method_naive_plain>(it->mp->vg, it->mp->vf, it->ip->instance, it->ip->g1_instance, 512, 32);
    hipDeviceSynchronize();
    libstl::resetlock_host(lockMem);

    hipEvent_t eventMSMStart, eventMSMEnd;
    hipEventCreate( &eventMSMStart);
	hipEventCreate( &eventMSMEnd);
    hipEventRecord( eventMSMStart, 0); 
    hipEventSynchronize(eventMSMStart);

    for(size_t i=0; i<1; i++)
    {
        it->res = libff::p_multi_exp_faster_multi_GPU_host<libff::G1<ppT>, libff::Fr<ppT>, libff::multi_exp_method_naive_plain>(it->mp->vg, it->mp->vf, it->ip->instance, it->ip->g1_instance, 512, 32);
        hipDeviceSynchronize();
    }

    hipEventRecord( eventMSMEnd, 0);
    hipEventSynchronize(eventMSMEnd);
    float   TimeMSM;
    hipEventElapsedTime( &TimeMSM, eventMSMStart, eventMSMEnd );
    printf( "Time thread %lu for MSM:  %3.5f ms\n", it->device_id, TimeMSM );

    return 0;
}

template<typename ppT_host, typename ppT_device>
void D2H(libff::G1<ppT_host>* hg1, libff::G1<ppT_device>* dg1, libff::G1<ppT_host>* g1_instance)
{
    hipMemcpy(hg1, dg1, sizeof(libff::G1<ppT_device>), hipMemcpyDeviceToHost);
    hg1->set_params(g1_instance->params);
}


template<typename ppT>
void Reduce(libff::G1<ppT>* hg1, libff::Fr<ppT>* instance, size_t total)
{
    int device_count;
    hipGetDeviceCount(&device_count);
    
    libff::G1<ppT> g1 = hg1[device_count-1];

    if(device_count != 1)
    {
        for(size_t i=device_count - 2; i <= device_count - 1; i--)
        {
            size_t log2_total = libff::log2(total);
            size_t c = log2_total - (log2_total / 3 - 2);
            size_t num_bits = instance->size_in_bits();
            size_t num_groups = (num_bits + c - 1) / c;
            size_t sgroup = (num_groups + device_count - 1) / device_count * i;
            size_t egroup = (num_groups + device_count - 1) / device_count * (i + 1);
            if(egroup > num_groups) egroup = num_groups;
            if(sgroup > num_groups) sgroup = num_groups;
            if(egroup == sgroup) continue;

            for(size_t j=0; j < (egroup - sgroup) * c; j++)
            {
                g1 = g1.dbl();
            }
            g1 = g1 + hg1[i];
        }
    }

    g1.to_special();

}

int main(int argc, char* argv[])
{
    if (argc < 2) {
		printf("Please enter the MSM scales (e.g. 20 represents 2^20) \n");
		return 1;
	}

    int log_size = atoi(argv[1]);

    int deviceCount;
    hipGetDeviceCount( &deviceCount );
    CUTThread  thread[deviceCount];
    printf("LOG: Number of devices: %d\n",deviceCount);
    printf("LOG: Starting to init public params\n");
    bls12_381_pp_host::init_public_params();
    printf("LOG: Done init public params\n");
    hipSetDevice(0);
    
    size_t num_v = (size_t) (1 << log_size);
    
    // params init 
    Mem device_mem[deviceCount];
    for(size_t i=0; i<deviceCount; i++)
    {
        device_mem[i].device_id = i;
        device_mem[i].mem = NULL;
        thread[i] = start_thread( multi_init_params, &device_mem[i] );
    }
    for(size_t i=0; i<deviceCount; i++)
    {
        end_thread(thread[i]);
    }

    // instance init
    instance_params* ip[deviceCount];
    Instance instance[deviceCount];
    for(size_t i=0; i<deviceCount; i++)
    {
        instance[i].device_id = i;
        instance[i].ip = &ip[i];
        thread[i] = start_thread( multi_instance_init, &instance[i] );
    }
    for(size_t i=0; i<deviceCount; i++)
    {
        end_thread(thread[i]);
    }

    h_instance_params hip;
    instance_init_host(&hip);

    printf("Log: Generating elements\n");
    // elements generation
    MSM_params<bls12_381_pp>* mp[deviceCount];
    for(size_t i=0; i<deviceCount; i++)
    {
        hipSetDevice(i);
        if( hipMalloc( (void**)&mp[i], sizeof(MSM_params<bls12_381_pp>)) != hipSuccess) printf("mp malloc error!\n");
        generate_MP<bls12_381_pp><<<1, 1>>>(mp[i], ip[i], num_v);
    }
    printf("LOG: Done Generating elements\n");
    for(size_t i=0; i<deviceCount; i++)
    {
        hipSetDevice(i);
        hipDeviceSynchronize();
    }
    hipSetDevice(0);
    
    // msm
    MSM<bls12_381_pp> msm[deviceCount];
    for(size_t i=0; i<deviceCount; i++)
    {
        msm[i].device_id = i;
        msm[i].mp = mp[i];
        msm[i].ip = ip[i];
        thread[i] = start_thread( multi_MSM<bls12_381_pp>, &msm[i] );
    }
    for(size_t i=0; i<deviceCount; i++)
    {
        end_thread(thread[i]);
    }

    libff::G1<bls12_381_pp_host> hg1[deviceCount];
    for(size_t i=0; i < deviceCount; i++)
    {
        hipSetDevice(i);
        D2H<bls12_381_pp_host, bls12_381_pp>(&hg1[i], msm[i].res, &hip.h_g1_instance);
    }

    Reduce<bls12_381_pp_host>(hg1, &hip.h_instance, num_v);

    hipDeviceReset();
    return 0;
}
